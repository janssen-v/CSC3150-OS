#include "hip/hip_runtime.h"
﻿#include "virtual_memory.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__device__ void init_invert_page_table(VirtualMemory *vm)
{
  for (int i = 0; i < vm->PAGE_ENTRIES; i++)
  {
    vm->invert_page_table[i] = 0x80000000; // invalid := MSB is 1
    vm->invert_page_table[i + vm->PAGE_ENTRIES] = i;
    // Page table limit is 1024 entries, since physical memory (data) only 32kb
    // 32kb/32 = 1024 pages (max) -> 10 bits
    // Remaining bits can be used to store LRU information with offset
  }
}

__device__ void vm_init(VirtualMemory *vm, uchar *buffer, uchar *storage,
                        u32 *invert_page_table, int *pagefault_num_ptr,
                        int PAGESIZE, int INVERT_PAGE_TABLE_SIZE,
                        int PHYSICAL_MEM_SIZE, int STORAGE_SIZE,
                        int PAGE_ENTRIES)
{
  // init variables
  vm->buffer = buffer;
  vm->storage = storage;
  vm->invert_page_table = invert_page_table;
  vm->pagefault_num_ptr = pagefault_num_ptr;

  // init constants
  vm->PAGESIZE = PAGESIZE;
  vm->INVERT_PAGE_TABLE_SIZE = INVERT_PAGE_TABLE_SIZE;
  vm->PHYSICAL_MEM_SIZE = PHYSICAL_MEM_SIZE;
  vm->STORAGE_SIZE = STORAGE_SIZE;
  vm->PAGE_ENTRIES = PAGE_ENTRIES;

  // before first vm_write or vm_read
  init_invert_page_table(vm);
}

// Additional function for updating page table
__device__ void lru_update(VirtualMemory *vm, int page_address)
{
  for (int i = 0; i < vm->PAGE_ENTRIES; i++)
  {
    int page_rank = vm->invert_page_table[i + vm->PAGE_ENTRIES];
    if (vm->invert_page_table[page_address + vm->PAGE_ENTRIES] < page_rank)
    {
      vm->invert_page_table[i + vm->PAGE_ENTRIES]--;
    }
  }
  vm->invert_page_table[page_address + vm->PAGE_ENTRIES] = vm->PAGE_ENTRIES - 1;
}

__device__ uchar vm_read(VirtualMemory *vm, u32 addr)
{
  /* Complete vm_read function to read single element from data buffer */
  u32 page_num = addr / vm->PAGESIZE;
  u32 offset = addr % vm->PAGESIZE;
  // Initialize address as invalid
  u32 memory_address = 0xFFFFFFFF;

  // Get physical address if it exists
  for (int i = 0; i < vm->PAGE_ENTRIES; i++)
  {
    if ((vm->invert_page_table[i]) == (page_num))
    {
      memory_address = (u32)i;
      break;
    }
  }

  // If in shared memory, update page page_rank (LRU)
  if (memory_address != 0xFFFFFFFF)
  {
    lru_update(vm, memory_address);
  }

  // Swap if not in shared memory
  else
  {
    // Get the least recently used page address
    u32 current_least = vm->invert_page_table[vm->PAGE_ENTRIES];
    u32 least_address = 0;
    for (int i = 0; i < vm->PAGE_ENTRIES; i++)
    {
      if (vm->invert_page_table[i + vm->PAGE_ENTRIES] < current_least)
      {
        current_least = vm->invert_page_table[i + vm->PAGE_ENTRIES];
        least_address = (u32)i;
      }
    }

    // Swap out least recently used page
    u32 LRU = vm->invert_page_table[least_address];
    for (int i = 0; i < vm->PAGESIZE; i++)
    {
      vm->storage[LRU * vm->PAGESIZE + i] = vm->buffer[least_address * vm->PAGESIZE + i];
      vm->buffer[least_address * vm->PAGESIZE + i] = vm->storage[page_num * vm->PAGESIZE + i];
    }

    // Increment pagefault number after swap
    atomicAdd(vm->pagefault_num_ptr, 1);
    lru_update(vm, least_address);
    vm->invert_page_table[least_address] = page_num;
    memory_address = least_address;
  }
  return vm->buffer[memory_address * vm->PAGESIZE + offset];
}

__device__ void vm_write(VirtualMemory *vm, u32 addr, uchar value)
{
  /* Complete vm_write function to write value into data buffer */
  u32 page_num = addr / vm->PAGESIZE;
  u32 offset = addr % vm->PAGESIZE;
  u32 physical_address = 0xFFFFFFFF;

  // Get physical address if it exists
  for (int i = 0; i < vm->PAGE_ENTRIES; i++)
  {
    if ((vm->invert_page_table[i]) == (page_num))
    {
      physical_address = (u32)i;
      break;
    }
  }

  // If in shared memory, write to address and update page rank (LRU)
  if (physical_address != 0xFFFFFFFF)
  {
    u32 address = physical_address * vm->PAGESIZE + offset;
    vm->buffer[address] = value;
    lru_update(vm, physical_address);
  }
  // If not in shared memory, swap or initialize before write
  else
  {   
    // Get the least recently used page address
    u32 current_least = vm->invert_page_table[vm->PAGE_ENTRIES];
    u32 least_address = 0;
    for (int i = 0; i < vm->PAGE_ENTRIES; i++)
    {
      if (vm->invert_page_table[i + vm->PAGE_ENTRIES] < current_least)
      {
        current_least = vm->invert_page_table[i + vm->PAGE_ENTRIES];
        least_address = (u32)i;
      }
    }

    // If LRU page is empty, write to it
    if (vm->invert_page_table[least_address] == 0x80000000)
    {
      vm->buffer[least_address * vm->PAGESIZE] = value;
      vm->invert_page_table[least_address] = page_num;
      atomicAdd(vm->pagefault_num_ptr, 1);
      lru_update(vm, least_address);
    }
    // If LRU page has existing content, evict the victim to storage
    else
    {
      // Swap out least recently used page
      u32 LRU = vm->invert_page_table[least_address];
      for (int i = 0; i < vm->PAGESIZE; i++)
      {
        vm->storage[LRU * vm->PAGESIZE + i] = vm->buffer[least_address * vm->PAGESIZE + i];
        vm->buffer[least_address * vm->PAGESIZE + i] = vm->storage[page_num * vm->PAGESIZE + i];
      }

      // Increment pagefault number after swap
      atomicAdd(vm->pagefault_num_ptr, 1);

      // Overwrite the evicted victim page
      vm->buffer[least_address * vm->PAGESIZE] = value;
      vm->invert_page_table[least_address] = page_num;

      // Update the pagetable & LRU
      lru_update(vm, least_address);
    }
  }
}

__device__ void vm_snapshot(VirtualMemory *vm, uchar *results, int offset,
                            int input_size)
{
  /* Complete snapshot function togther with vm_read to load elements from data
   * to result buffer */
  for (int i = 0; i < input_size; i++)
  {
    results[offset + i] = vm_read(vm, i);
  }
}
